#include "hip/hip_runtime.h"
/*Crown Copyright 2012 AWE.
 *
 * This file is part of CloverLeaf.
 *
 * CloverLeaf is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * CloverLeaf is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * CloverLeaf. If not, see http://www.gnu.org/licenses/.
 */

/*
 *  @brief CUDA revert kernel.
 *  @author Michael Boulton NVIDIA Corporation
 *  @details Takes the half step field data used in the predictor and reverts
 *  it to the start of step data, ready for the corrector.
 *  Note that this does not seem necessary in this proxy-app but should be
 *  left in to remain relevant to the full method.
 */


#include <iostream>
#include "ftocmacros.h"
#include "cuda_common.cu"

#include "chunk_cuda.cu"
extern CloverleafCudaChunk chunk;

__global__ void device_revert_kernel_cuda
(int x_min, int x_max, int y_min, int y_max,
const double* __restrict const density0,
      double* __restrict const density1,
const double* __restrict const energy0,
      double* __restrict const energy1)
{
    __kernel_indexes;

    if (row >= (y_min + 1) && row <= (y_max + 1)
    && column >= (x_min + 1) && column <= (x_max + 1))
    {
        density1[THARR2D(0, 0, 0)] = density0[THARR2D(0, 0, 0)];
        energy1[THARR2D(0, 0, 0)] = energy0[THARR2D(0, 0, 0)];
    }
}

extern "C" void revert_kernel_cuda_
(void)
{
    chunk.revert_kernel();
}

void CloverleafCudaChunk::revert_kernel
(void)
{
    CUDA_BEGIN_PROFILE;

    device_revert_kernel_cuda<<< num_blocks, BLOCK_SZ >>>
    (x_min,x_max,y_min,y_max, density0, density1, energy0, energy1);
    CUDA_ERR_CHECK;

    CUDA_END_PROFILE;
}

