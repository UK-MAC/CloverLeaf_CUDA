#include "hip/hip_runtime.h"
/*Crown Copyright 2012 AWE.
 *
 * This file is part of CloverLeaf.
 *
 * CloverLeaf is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * CloverLeaf is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * CloverLeaf. If not, see http://www.gnu.org/licenses/.
 */

/*
 *  @brief CUDA common file
 *  @author Michael Boulton NVIDIA Corporation
 *  @details Contains common elements for cuda kernels
 */

#ifndef __CUDA_COMMON_INC
#define __CUDA_COMMON_INC

// size of workgroup/block - 256 seems to be optimal
#ifndef BLOCK_SZ 
    #define BLOCK_SZ 256
#endif

// number of bytes to allocate for x size array
#define BUFSZX(x_extra)   \
    ( ((x_max) + 4 + x_extra)       \
    * sizeof(double) )

// number of bytes to allocate for y size array
#define BUFSZY(y_extra)   \
    ( ((y_max) + 4 + y_extra)       \
    * sizeof(double) )

// number of bytes to allocate for 2d array
#define BUFSZ2D(x_extra, y_extra)   \
    ( ((x_max) + 4 + x_extra)       \
    * ((y_max) + 4 + y_extra)       \
    * sizeof(double) )

/*
*  access a value in a 2d array given the x and y offset from current thread
*  index, adding or subtracting a bit more if it is one of the arrays with
*  bigger rows
*/
#define THARR2D(x_offset, y_offset, big_row)\
    ( glob_id                               \
    + (x_offset)                            \
    + ((y_offset) * (x_max + 4))            \
    + (big_row * (row + (y_offset))) )

// kernel indexes uses in all kernels
#define __kernel_indexes                    \
    const int glob_id = threadIdx.x         \
        + blockIdx.x * blockDim.x;          \
    const int row = glob_id / (x_max + 4);  \
    const int column = glob_id % (x_max + 4);

// used in update_halo and for copying back to host for mpi transfers
#define FIELD_density0      0
#define FIELD_density1      1
#define FIELD_energy0       2
#define FIELD_energy1       3
#define FIELD_pressure      4
#define FIELD_viscosity     5
#define FIELD_soundspeed    6
#define FIELD_xvel0         7
#define FIELD_xvel1         8
#define FIELD_yvel0         9
#define FIELD_yvel1         10
#define FIELD_vol_flux_x    11
#define FIELD_vol_flux_y    12
#define FIELD_mass_flux_x   13
#define FIELD_mass_flux_y   14
#define NUM_FIELDS          15

enum {CELL_DATA, VERTEX_DATA, X_FACE_DATA, Y_FACE_DATA};

/*******************/

// disable checking for errors after kernel calls / memory allocation
#ifdef NO_ERR_CHK

// do nothing instead
#define CUDA_ERR_CHECK ;

#else

#include <iostream>

#define CUDA_ERR_CHECK errorHandler(__LINE__, __FILE__);

static const char* errorCodes
(int err_code)
{
    switch(err_code)
    {
        case hipSuccess: return "hipSuccess";
        case hipErrorMissingConfiguration: return "hipErrorMissingConfiguration";
        case hipErrorOutOfMemory: return "hipErrorOutOfMemory";
        case hipErrorNotInitialized: return "hipErrorNotInitialized";
        case hipErrorLaunchFailure: return "hipErrorLaunchFailure";
        case hipErrorPriorLaunchFailure: return "hipErrorPriorLaunchFailure";
        case hipErrorLaunchTimeOut: return "hipErrorLaunchTimeOut";
        case hipErrorLaunchOutOfResources: return "hipErrorLaunchOutOfResources";
        case hipErrorInvalidDeviceFunction: return "hipErrorInvalidDeviceFunction";
        case hipErrorInvalidConfiguration: return "hipErrorInvalidConfiguration";
        case hipErrorInvalidDevice: return "hipErrorInvalidDevice";
        case hipErrorInvalidValue: return "hipErrorInvalidValue";
        case hipErrorInvalidPitchValue: return "hipErrorInvalidPitchValue";
        case hipErrorInvalidSymbol: return "hipErrorInvalidSymbol";
        case hipErrorMapFailed: return "hipErrorMapFailed";
        case hipErrorUnmapFailed: return "hipErrorUnmapFailed";
        case cudaErrorInvalidHostPointer: return "cudaErrorInvalidHostPointer";
        case hipErrorInvalidDevicePointer: return "hipErrorInvalidDevicePointer";
        case hipErrorInvalidTexture: return "hipErrorInvalidTexture";
        case cudaErrorInvalidTextureBinding: return "cudaErrorInvalidTextureBinding";
        case hipErrorInvalidChannelDescriptor: return "hipErrorInvalidChannelDescriptor";
        case hipErrorInvalidMemcpyDirection: return "hipErrorInvalidMemcpyDirection";
        case cudaErrorAddressOfConstant: return "cudaErrorAddressOfConstant";
        case cudaErrorTextureFetchFailed: return "cudaErrorTextureFetchFailed";
        case cudaErrorTextureNotBound: return "cudaErrorTextureNotBound";
        case cudaErrorSynchronizationError: return "cudaErrorSynchronizationError";
        case cudaErrorInvalidFilterSetting: return "cudaErrorInvalidFilterSetting";
        case cudaErrorInvalidNormSetting: return "cudaErrorInvalidNormSetting";
        case cudaErrorMixedDeviceExecution: return "cudaErrorMixedDeviceExecution";
        case hipErrorDeinitialized: return "hipErrorDeinitialized";
        case hipErrorUnknown: return "hipErrorUnknown";
        case cudaErrorNotYetImplemented: return "cudaErrorNotYetImplemented";
        case cudaErrorMemoryValueTooLarge: return "cudaErrorMemoryValueTooLarge";
        case hipErrorInvalidHandle: return "hipErrorInvalidHandle";
        case hipErrorNotReady: return "hipErrorNotReady";
        case hipErrorInsufficientDriver: return "hipErrorInsufficientDriver";
        case hipErrorSetOnActiveProcess: return "hipErrorSetOnActiveProcess";
        case cudaErrorInvalidSurface: return "cudaErrorInvalidSurface";
        case hipErrorNoDevice: return "hipErrorNoDevice";
        case hipErrorECCNotCorrectable: return "hipErrorECCNotCorrectable";
        case hipErrorSharedObjectSymbolNotFound: return "hipErrorSharedObjectSymbolNotFound";
        case hipErrorSharedObjectInitFailed: return "hipErrorSharedObjectInitFailed";
        case hipErrorUnsupportedLimit: return "hipErrorUnsupportedLimit";
        case cudaErrorDuplicateVariableName: return "cudaErrorDuplicateVariableName";
        case cudaErrorDuplicateTextureName: return "cudaErrorDuplicateTextureName";
        case cudaErrorDuplicateSurfaceName: return "cudaErrorDuplicateSurfaceName";
        case cudaErrorDevicesUnavailable: return "cudaErrorDevicesUnavailable";
        case hipErrorInvalidImage: return "hipErrorInvalidImage";
        case hipErrorNoBinaryForGpu: return "hipErrorNoBinaryForGpu";
        case cudaErrorIncompatibleDriverContext: return "cudaErrorIncompatibleDriverContext";
        case hipErrorPeerAccessAlreadyEnabled: return "hipErrorPeerAccessAlreadyEnabled";
        case hipErrorPeerAccessNotEnabled: return "hipErrorPeerAccessNotEnabled";
        case hipErrorContextAlreadyInUse: return "hipErrorContextAlreadyInUse";
        case hipErrorProfilerDisabled: return "hipErrorProfilerDisabled";
        case hipErrorProfilerNotInitialized: return "hipErrorProfilerNotInitialized";
        case hipErrorProfilerAlreadyStarted: return "hipErrorProfilerAlreadyStarted";
        case hipErrorProfilerAlreadyStopped: return "hipErrorProfilerAlreadyStopped";
        case hipErrorAssert: return "hipErrorAssert";
        case cudaErrorTooManyPeers: return "cudaErrorTooManyPeers";
        case hipErrorHostMemoryAlreadyRegistered: return "hipErrorHostMemoryAlreadyRegistered";
        case hipErrorHostMemoryNotRegistered: return "hipErrorHostMemoryNotRegistered";
        case hipErrorOperatingSystem: return "hipErrorOperatingSystem";
        case cudaErrorStartupFailure: return "cudaErrorStartupFailure";
        case cudaErrorApiFailureBase: return "cudaErrorApiFailureBase";
        default: return "Unknown error";
    }
}

inline void errorHandler
(int line_num, std::string const& file)
{
    hipDeviceSynchronize();
    int l_e = hipGetLastError();
    if (hipSuccess != l_e)
    {
        std::cout << "error on line " << line_num << " of ";
        std::cout << file << std::endl;
        std::cout << "return code " << l_e; 
        std::cout << " (" << errorCodes(l_e) << ")";
        std::cout << std::endl;
        exit(l_e);
    }
}

#endif //NO_ERR_CHK

// whether to time kernel run times
#ifdef TIME_KERNELS

// beginning of profiling bit
#define CUDA_BEGIN_PROFILE \
    double __t_0, __t_1;          \
    __t_0 = MPI_Wtime();

// end of profiling bit
#define CUDA_END_PROFILE \
    hipDeviceSynchronize();                        \
    __t_1 = MPI_Wtime();                       \
    std::cout << "[PROFILING] " << __t_1 - __t_0  \
    << " to calculate " << __FILE__  << std::endl;

#else

#define CUDA_BEGIN_PROFILE ;
#define CUDA_END_PROFILE ;

#endif // TIME_KERNELS

typedef struct cell_info {
    const int x_e;
    const int y_e;
    const int x_i;
    const int y_i;
    const int x_f;
    const int y_f;
    const int grid_type;

    cell_info
    (int x_extra, int y_extra,
    int x_invert, int y_invert,
    int x_face, int y_face,
    int in_type)
    :x_e(x_extra), y_e(y_extra),
    x_i(x_invert), y_i(y_invert),
    x_f(x_face), y_f(y_face),
    grid_type(in_type)
    {
        ;
    }

} cell_info_t;

// types of array data
const static cell_info_t CELL(    0, 0,  1,  1, 0, 0, CELL_DATA);
const static cell_info_t VERTEX_X(1, 1, -1,  1, 0, 0, VERTEX_DATA);
const static cell_info_t VERTEX_Y(1, 1,  1, -1, 0, 0, VERTEX_DATA);
const static cell_info_t X_FACE(  1, 0, -1,  1, 1, 0, X_FACE_DATA);
const static cell_info_t Y_FACE(  0, 1,  1, -1, 0, 1, Y_FACE_DATA);

#include "ftocmacros.h"

// callbacks for reductions
__device__ inline static int sum_func (int x, int y) { return x + y; }
__device__ inline static int min_func (int x, int y) { return MIN(x, y); }
__device__ inline static int max_func (int x, int y) { return MAX(x, y); }

__device__ inline static double sum_func (double x, double y) { return x + y; }
__device__ inline static double min_func (double x, double y) { return MIN(x, y); }
__device__ inline static double max_func (double x, double y) { return MAX(x, y); }

template < int offset >
class Reduce
{
public:
    __device__ inline static void run
    (double* array, double* out, double(*func)(double, double))
    {
        // only need to synch if not working within a warp
        if (offset > 16)
        {
            __syncthreads();
        }

        // only continue if it's in the lower half
        if (threadIdx.x < offset)
        {
            array[threadIdx.x] = func(array[threadIdx.x], array[threadIdx.x + offset]);
            Reduce< offset/2 >::run(array, out, func);
        }
    }

    __device__ inline static void run
    (int* array, int* out, int(*func)(int, int))
    {
        // only need to synch if not working within a warp
        if (offset > 16)
        {
            __syncthreads();
        }

        // only continue if it's in the lower half
        if (threadIdx.x < offset)
        {
            array[threadIdx.x] = func(array[threadIdx.x], array[threadIdx.x + offset]);
            Reduce< offset/2 >::run(array, out, func);
        }
    }
};

template < >
class Reduce < 0 >
{
public:
    __device__ inline static void run
    (double* array, double* out, double(*func)(double, double))
    {
        out[blockIdx.x] = array[0];
    }

    __device__ inline static void run
    (int* array, int* out, int(*func)(int, int))
    {
        out[blockIdx.x] = array[0];
    }
};

#endif

