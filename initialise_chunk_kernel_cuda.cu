#include "hip/hip_runtime.h"
/*Crown Copyright 2012 AWE.
 *
 * This file is part of CloverLeaf.
 *
 * CloverLeaf is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * CloverLeaf is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * CloverLeaf. If not, see http://www.gnu.org/licenses/.
 */

/*
 *  @brief CUDA driver for chunk initialisation.
 *  @author Michael Boulton NVIDIA Corporation
 *  @details Invokes the user specified chunk initialisation kernel.
 */


#include <iostream>
#include "ftocmacros.h"
#include "cuda_common.cu"

#include "chunk_cuda.cu"
extern CloverleafCudaChunk chunk;

__global__ void device_initialise_chunk_kernel_vertex_cuda
(int x_min, int x_max, int y_min, int y_max,
const double d_xmin,
const double d_ymin,
const double d_dx,
const double d_dy,
double* __restrict const vertexx,
double* __restrict const vertexdx,
double* __restrict const vertexy,
double* __restrict const vertexdy)
{
    const int glob_id = threadIdx.x
        + blockIdx.x * blockDim.x;

    //bigger indexes 
    const int row = glob_id / (x_max + 5);
    const int column = glob_id % (x_max + 5);

    //fill out x arrays
    if (row == 0)
    {
        vertexx[column] = d_xmin + d_dx
            * static_cast<double>((column - 1) - x_min);
        vertexdx[column] = d_dx;
    }

    // fill out y arrays
    if (column == 0)
    {
        vertexy[row] = d_ymin + d_dy
            * static_cast<double>((row - 1) - y_min);
        vertexdy[row] = d_dy;
    }

}

__global__ void device_initialise_chunk_kernel_cuda
(int x_min, int x_max, int y_min, int y_max,
const double d_xmin,
const double d_ymin,
const double d_dx,
const double d_dy,
const double* __restrict const vertexx,
const double* __restrict const vertexdx,
const double* __restrict const vertexy,
const double* __restrict const vertexdy,
      double* __restrict const cellx,
      double* __restrict const celldx,
      double* __restrict const celly,
      double* __restrict const celldy,
      double* __restrict const volume, 
      double* __restrict const xarea, 
      double* __restrict const yarea)
{
    __kernel_indexes;

    //fill x arrays
    if (row == 0)
    {
        cellx[column] = 0.5 * (vertexx[column] + vertexx[column + 1]);
        celldx[column] = d_dx;
    }

    //fill y arrays
    if (column == 0)
    {
        celly[row] = 0.5 * (vertexy[row] + vertexy[row + 1]);
        celldy[row] = d_dy;
    }

    if (row >= (y_min + 1) - 2 && row <= (y_max + 1) + 2
    && column >= (x_min + 1) - 2 && column <= (x_max + 1) + 2)
    {
        volume[THARR2D(0, 0, 0)] = d_dx * d_dy;
        xarea[THARR2D(0, 0, 1)] = d_dy;
        yarea[THARR2D(0, 0, 0)] = d_dx;
    }

}

extern "C" void initialise_chunk_kernel_cuda_
(double* d_xmin, double* d_ymin, double* d_dx, double* d_dy)
{
    chunk.initialise_chunk_kernel(*d_xmin, *d_ymin, *d_dx, *d_dy);
}

void CloverleafCudaChunk::initialise_chunk_kernel
(double d_xmin, double d_ymin, double d_dx, double d_dy)
{
    CUDA_BEGIN_PROFILE;

    device_initialise_chunk_kernel_vertex_cuda<<< num_blocks, BLOCK_SZ >>>
    (x_min,x_max,y_min,y_max, d_xmin, d_ymin, d_dx, d_dy, 
        vertexx, vertexdx, vertexy, vertexdy);
    CUDA_ERR_CHECK;

    device_initialise_chunk_kernel_cuda<<< num_blocks, BLOCK_SZ >>>
    (x_min,x_max,y_min,y_max, d_xmin, d_ymin, d_dx, d_dy, 
        vertexx, vertexdx, vertexy, vertexdy,
        cellx, celldx, celly, celldy,
        volume, xarea, yarea);
    CUDA_ERR_CHECK;

    CUDA_END_PROFILE;
}

