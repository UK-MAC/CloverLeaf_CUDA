#include "hip/hip_runtime.h"

#include <iostream>
#include "cuda_common.cu"
#include "ftocmacros.h"
#include <algorithm>

#include "chunk_cuda.cu"
extern CloverleafCudaChunk chunk;

#include "omp.h"

extern CudaDevPtrStorage pointer_storage;

__global__ void device_PdV_cuda_kernel_predict
(int x_min, int x_max, int y_min, int y_max, 
double dt,
         int * __restrict const error_condition,
const double * __restrict const xarea,
const double * __restrict const yarea,
const double * __restrict const volume,
const double * __restrict const density0,
      double * __restrict const density1,
const double * __restrict const energy0,
      double * __restrict const energy1,
const double * __restrict const pressure,
const double * __restrict const viscosity,
const double * __restrict const xvel0,
const double * __restrict const yvel0,
const double * __restrict const xvel1,
const double * __restrict const yvel1)
{
    __kernel_indexes;

    __shared__ int err_cond_kernel[BLOCK_SZ];
    err_cond_kernel[threadIdx.x] = 0;

    double volume_change;
    double recip_volume, energy_change, min_cell_volume,
        right_flux, left_flux, top_flux, bottom_flux, total_flux;
    
    if(row > 1 && column > 1
    && row < y_max+2 && column < x_max+2)
    {
        left_flux   = (xarea[THARR2D(0, 0, 1)]
            * (xvel0[THARR2D(0, 0, 1)] + xvel0[THARR2D(0, 0, 1)] 
            + xvel0[THARR2D(0, 1, 1)] + xvel0[THARR2D(0, 1, 1)]))
            * 0.25 * dt * 0.5;
        right_flux  = (xarea[THARR2D(1, 0, 1)]
            * (xvel0[THARR2D(1, 0, 1)] + xvel0[THARR2D(1, 0, 1)] 
            + xvel0[THARR2D(1, 1, 1)] + xvel0[THARR2D(1, 1, 1)]))
            * 0.25 * dt * 0.5;

        bottom_flux = (yarea[THARR2D(0, 0, 0)]
            * (yvel0[THARR2D(0, 0, 1)] + yvel0[THARR2D(0, 0, 1)] 
            + yvel0[THARR2D(1, 0, 1)] + yvel0[THARR2D(1, 0, 1)]))
            * 0.25 * dt * 0.5;
        top_flux    = (yarea[THARR2D(0, 1, 0)]
            * (yvel0[THARR2D(0, 1, 1)] + yvel0[THARR2D(0, 1, 1)] 
            + yvel0[THARR2D(1, 1, 1)] + yvel0[THARR2D(1, 1, 1)]))
            * 0.25 * dt * 0.5;

        total_flux = right_flux - left_flux + top_flux - bottom_flux;

        volume_change = volume[THARR2D(0, 0, 0)]
            / (volume[THARR2D(0, 0, 0)] + total_flux);

        //minimum of total, horizontal, and vertical flux
        min_cell_volume = 
            MIN(volume[THARR2D(0, 0, 0)] + total_flux,
            MIN(volume[THARR2D(0, 0, 0)] + right_flux - left_flux,
                volume[THARR2D(0, 0, 0)] + top_flux - bottom_flux));

        if(volume_change <= 0.0)
        {
            err_cond_kernel[threadIdx.x] = 1;
        }
        if(min_cell_volume <= 0.0)
        {
            err_cond_kernel[threadIdx.x] = 2;
        }

        recip_volume = 1.0/volume[THARR2D(0, 0, 0)];

        energy_change = ((pressure[THARR2D(0, 0, 0)] / density0[THARR2D(0, 0, 0)])
            + (viscosity[THARR2D(0, 0, 0)] / density0[THARR2D(0, 0, 0)]))
            * total_flux * recip_volume;

        energy1[THARR2D(0, 0, 0)] = energy0[THARR2D(0, 0, 0)] - energy_change;
        density1[THARR2D(0, 0, 0)] = density0[THARR2D(0, 0, 0)] * volume_change;
    }

    //reduction to get error conditon, if any
    __syncthreads();
    for(size_t offset = blockDim.x / 2; offset > 0; offset /= 2)
    {
        if(threadIdx.x < offset)
        {
            err_cond_kernel[threadIdx.x] = MAX(err_cond_kernel[threadIdx.x],
                err_cond_kernel[threadIdx.x + offset]);
        }
        __syncthreads();
    }
    error_condition[blockIdx.x] = err_cond_kernel[0];;
}

__global__ void device_PdV_cuda_kernel_not_predict
(int x_min, int x_max, int y_min, int y_max, 
double dt,
         int * __restrict const error_condition,
const double * __restrict const xarea,
const double * __restrict const yarea,
const double * __restrict const volume,
const double * __restrict const density0,
      double * __restrict const density1,
const double * __restrict const energy0,
      double * __restrict const energy1,
const double * __restrict const pressure,
const double * __restrict const viscosity,
const double * __restrict const xvel0,
const double * __restrict const yvel0,
const double * __restrict const xvel1,
const double * __restrict const yvel1)
{
    __kernel_indexes;

    __shared__ int err_cond_kernel[BLOCK_SZ];
    err_cond_kernel[threadIdx.x] = 0;

    double volume_change;
    double recip_volume, energy_change, min_cell_volume,
        right_flux, left_flux, top_flux, bottom_flux, total_flux;
    
    if(row > 1 && column > 1
    && row < y_max+2 && column < x_max+2)
    {
        left_flux   = (xarea[THARR2D(0, 0, 1)]
            * (xvel0[THARR2D(0, 0, 1)] + xvel1[THARR2D(0, 0, 1)] 
            + xvel0[THARR2D(0, 1, 1)] + xvel1[THARR2D(0, 1, 1)]))
            * 0.25 * dt;
        right_flux  = (xarea[THARR2D(1, 0, 1)]
            * (xvel0[THARR2D(1, 0, 1)] + xvel1[THARR2D(1, 0, 1)] 
            + xvel0[THARR2D(1, 1, 1)] + xvel1[THARR2D(1, 1, 1)]))
            * 0.25 * dt;

        bottom_flux = (yarea[THARR2D(0, 0, 0)]
            * (yvel0[THARR2D(0, 0, 1)] + yvel1[THARR2D(0, 0, 1)] 
            + yvel0[THARR2D(1, 0, 1)] + yvel1[THARR2D(1, 0, 1)]))
            * 0.25 * dt;
        top_flux    = (yarea[THARR2D(0, 1, 0)]
            * (yvel0[THARR2D(0, 1, 1)] + yvel1[THARR2D(0, 1, 1)] 
            + yvel0[THARR2D(1, 1, 1)] + yvel1[THARR2D(1, 1, 1)]))
            * 0.25 * dt;

        total_flux = right_flux - left_flux + top_flux - bottom_flux;

        volume_change = volume[THARR2D(0, 0, 0)]
            / (volume[THARR2D(0, 0, 0)] + total_flux);

        min_cell_volume =
            MIN(volume[THARR2D(0, 0, 0)] + total_flux,
            MIN(volume[THARR2D(0, 0, 0)] + right_flux - left_flux,
                volume[THARR2D(0, 0, 0)] + top_flux - bottom_flux));

        if(volume_change <= 0.0)
        {
            err_cond_kernel[threadIdx.x] = 1;
        }
        if(min_cell_volume <= 0.0)
        {
            err_cond_kernel[threadIdx.x] = 2;
        }

        recip_volume = 1.0/volume[THARR2D(0, 0, 0)];

        energy_change = ((pressure[THARR2D(0, 0, 0)] / density0[THARR2D(0, 0, 0)])
            + (viscosity[THARR2D(0, 0, 0)] / density0[THARR2D(0, 0, 0)]))
            * total_flux * recip_volume;

        energy1[THARR2D(0, 0, 0)] = energy0[THARR2D(0, 0, 0)] - energy_change;
        density1[THARR2D(0, 0, 0)] = density0[THARR2D(0, 0, 0)] * volume_change;

    }

    __syncthreads();
    for(size_t offset = blockDim.x / 2; offset > 0; offset /= 2)
    {
        if(threadIdx.x < offset)
        {
            err_cond_kernel[threadIdx.x] = MAX(err_cond_kernel[threadIdx.x],
                err_cond_kernel[threadIdx.x + offset]);
        }
        __syncthreads();
    }
    error_condition[blockIdx.x] = err_cond_kernel[0];;
}

void PdV_cuda
(int error_condition,int predict,int x_min,int x_max,int y_min,int y_max,
double dt,
double *xarea,
double *yarea,
double *volume,
double *density0,
double *density1,
double *energy0,
double *energy1,
double *pressure,
double *viscosity,
double *xvel0,
double *yvel0,
double *xvel1,
double *yvel1)
{
    pointer_storage.setSize(x_max, y_max);

    double* xarea_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, xarea, BUFSZ2D(1, 0));
    double* yarea_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, yarea, BUFSZ2D(0, 1));

    double* density0_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, density0, BUFSZ2D(0, 0));
    double* energy0_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, energy0, BUFSZ2D(0, 0));

    double* volume_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, volume, BUFSZ2D(0, 0));
    double* pressure_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, pressure, BUFSZ2D(0, 0));
    double* viscosity_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, viscosity, BUFSZ2D(0, 0));

    double* xvel0_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, xvel0, BUFSZ2D(1, 1));
    double* xvel1_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, xvel1, BUFSZ2D(1, 1));
    double* yvel0_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, yvel0, BUFSZ2D(1, 1));
    double* yvel1_d = pointer_storage.getDevStorageAndCopy(__LINE__, __FILE__, yvel1, BUFSZ2D(1, 1));

    double* energy1_d = pointer_storage.getDevStorage(__LINE__, __FILE__);
    double* density1_d = pointer_storage.getDevStorage(__LINE__, __FILE__);

    size_t num_blocks = (((x_max+4)*(y_max+4))/BLOCK_SZ);
    //error condition
    thrust::device_ptr<int> reduce_ptr_1 =
        thrust::device_malloc<int>(num_blocks*sizeof(int));
    int* err_condition_arr_d = thrust::raw_pointer_cast(reduce_ptr_1);

#ifdef TIME_KERNELS
_CUDA_BEGIN_PROFILE_name(device);
#endif

    if(predict)
    {
        device_PdV_cuda_kernel_predict<<< ((x_max+4)*(y_max+4))/BLOCK_SZ, BLOCK_SZ >>>
        (x_min, x_max, y_min, y_max, dt, err_condition_arr_d,
            xarea_d, yarea_d, volume_d, density0_d, density1_d,
            energy0_d, energy1_d, pressure_d, viscosity_d,
            xvel0_d, yvel0_d, xvel1_d, yvel1_d);
    }
    else
    {
        device_PdV_cuda_kernel_not_predict<<< ((x_max+4)*(y_max+4))/BLOCK_SZ, BLOCK_SZ >>>
        (x_min, x_max, y_min, y_max, dt, err_condition_arr_d,
            xarea_d, yarea_d, volume_d, density0_d, density1_d,
            energy0_d, energy1_d, pressure_d, viscosity_d,
            xvel0_d, yvel0_d, xvel1_d, yvel1_d);
    }

#ifdef TIME_KERNELS
_CUDA_END_PROFILE_name(device);
#endif

errChk(__LINE__, __FILE__);

    pointer_storage.freeDevStorageAndCopy(energy1_d, energy1, BUFSZ2D(0, 0));
    pointer_storage.freeDevStorageAndCopy(density1_d, density1, BUFSZ2D(0, 0));

    pointer_storage.freeDevStorage(xarea_d);
    pointer_storage.freeDevStorage(yarea_d);
    pointer_storage.freeDevStorage(volume_d);
    pointer_storage.freeDevStorage(pressure_d);
    pointer_storage.freeDevStorage(viscosity_d);
    pointer_storage.freeDevStorage(xvel0_d);
    pointer_storage.freeDevStorage(yvel0_d);
    pointer_storage.freeDevStorage(xvel1_d);
    pointer_storage.freeDevStorage(yvel1_d);
    pointer_storage.freeDevStorage(energy0_d);
    pointer_storage.freeDevStorage(density0_d);

    /*
    int err_cond = thrust::reduce(reduce_ptr_1,
        reduce_ptr_1 + num_blocks,
        0, thrust::maximum<int>());
    // */
    int err_cond = *thrust::max_element(reduce_ptr_1, reduce_ptr_1 + num_blocks);
    thrust::device_free(reduce_ptr_1);

    if(err_cond == 1)
    {
        std::cerr << "Negative volume in PdV kernel" << std::endl;
    }
    else if(err_cond == 2)
    {
        std::cerr << "Negative cell volume in PdV kernel" << std::endl;
    }

}

extern "C" void pdv_kernel_cuda_
(int *errorcondition,int *prdct,
int *xmin,int *xmax,int *ymin,int *ymax,double *dtbyt,
double *xarea,double *yarea,double *volume,
double *density0,
double *density1,
double *energy0,
double *energy1,
double *pressure,
double *viscosity,
double *xvel0,
double *xvel1,
double *yvel0,
double *yvel1)
{
#ifdef TIME_KERNELS
_CUDA_BEGIN_PROFILE_name(host);
#endif
    #ifndef CUDA_RESIDENT
    PdV_cuda(*errorcondition, *prdct,  *xmin, *xmax, *ymin, *ymax,*dtbyt,
        xarea, yarea, volume, density0, density1, energy0, energy1,
        pressure, viscosity, xvel0, yvel0, xvel1, yvel1);
    #else
    chunk.PdV_kernel(errorcondition, *prdct, *dtbyt);
    #endif
#ifdef TIME_KERNELS
_CUDA_END_PROFILE_name(host);
#endif
}


void CloverleafCudaChunk::PdV_kernel
(int* error_condition, int predict, double dt)
{

#ifdef TIME_KERNELS
_CUDA_BEGIN_PROFILE_name(device);
#endif

    if(predict)
    {
        device_PdV_cuda_kernel_predict<<< num_blocks, BLOCK_SZ >>>
        (x_min, x_max, y_min, y_max, dt, pdv_reduce_array,
            xarea, yarea, volume, density0, density1,
            energy0, energy1, pressure, viscosity,
            xvel0, yvel0, xvel1, yvel1);
        errChk(__LINE__, __FILE__);
    }
    else
    {
        device_PdV_cuda_kernel_not_predict<<< num_blocks, BLOCK_SZ >>>
        (x_min, x_max, y_min, y_max, dt, pdv_reduce_array,
            xarea, yarea, volume, density0, density1,
            energy0, energy1, pressure, viscosity,
            xvel0, yvel0, xvel1, yvel1);
        errChk(__LINE__, __FILE__);
    }

#ifdef TIME_KERNELS
_CUDA_END_PROFILE_name(device);
#endif

    int err_cond = *thrust::max_element(reduce_pdv,
        reduce_pdv + num_blocks);

    if(err_cond == 1)
    {
        std::cerr << "Negative volume in PdV kernel" << std::endl;
    }
    else if(err_cond == 2)
    {
        std::cerr << "Negative cell volume in PdV kernel" << std::endl;
    }
}

