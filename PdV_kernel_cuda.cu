#include "hip/hip_runtime.h"
/*Crown Copyright 2012 AWE.
 *
 * This file is part of CloverLeaf.
 *
 * CloverLeaf is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the
 * Free Software Foundation, either version 3 of the License, or (at your option)
 * any later version.
 *
 * CloverLeaf is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 * You should have received a copy of the GNU General Public License along with
 * CloverLeaf. If not, see http://www.gnu.org/licenses/.
 */

/*
 *  @brief CUDA PdV kernel.
 *  @author Michael Boulton NVIDIA Corporation
 *  @details Calculates the change in energy and density in a cell using the
 *  change on cell volume due to the velocity gradients in a cell. The time
 *  level of the velocity data depends on whether it is invoked as the
 *  predictor or corrector.
 */

#include <iostream>
#include "cuda_common.cu"
#include "ftocmacros.h"

#include <algorithm>

#include "chunk_cuda.cu"
extern CloverleafCudaChunk chunk;

__global__ void device_PdV_cuda_kernel_predict
(int x_min, int x_max, int y_min, int y_max, double dt,
         int * __restrict const error_condition,
const double * __restrict const xarea,
const double * __restrict const yarea,
const double * __restrict const volume,
const double * __restrict const density0,
      double * __restrict const density1,
const double * __restrict const energy0,
      double * __restrict const energy1,
const double * __restrict const pressure,
const double * __restrict const viscosity,
const double * __restrict const xvel0,
const double * __restrict const yvel0,
const double * __restrict const xvel1,
const double * __restrict const yvel1)
{
    __kernel_indexes;

    __shared__ int err_condition_shared[BLOCK_SZ];
    err_condition_shared[threadIdx.x] = 0;

    double volume_change;
    double recip_volume, energy_change, min_cell_volume,
        right_flux, left_flux, top_flux, bottom_flux, total_flux;
    
    if (row >= (y_min + 1) && row <= (y_max + 1)
    && column >= (x_min + 1) && column <= (x_max + 1))
    {
        left_flux   = (xarea[THARR2D(0, 0, 1)]
            * (xvel0[THARR2D(0, 0, 1)] + xvel0[THARR2D(0, 1, 1)] 
            + xvel0[THARR2D(0, 0, 1)] + xvel0[THARR2D(0, 1, 1)]))
            * 0.25 * dt * 0.5;
        right_flux  = (xarea[THARR2D(1, 0, 1)]
            * (xvel0[THARR2D(1, 0, 1)] + xvel0[THARR2D(1, 1, 1)] 
            + xvel0[THARR2D(1, 0, 1)] + xvel0[THARR2D(1, 1, 1)]))
            * 0.25 * dt * 0.5;

        bottom_flux = (yarea[THARR2D(0, 0, 0)]
            * (yvel0[THARR2D(0, 0, 1)] + yvel0[THARR2D(1, 0, 1)] 
            + yvel0[THARR2D(0, 0, 1)] + yvel0[THARR2D(1, 0, 1)]))
            * 0.25 * dt * 0.5;
        top_flux    = (yarea[THARR2D(0, 1, 0)]
            * (yvel0[THARR2D(0, 1, 1)] + yvel0[THARR2D(1, 1, 1)] 
            + yvel0[THARR2D(0, 1, 1)] + yvel0[THARR2D(1, 1, 1)]))
            * 0.25 * dt * 0.5;

        total_flux = right_flux - left_flux + top_flux - bottom_flux;

        volume_change = volume[THARR2D(0, 0, 0)]
            / (volume[THARR2D(0, 0, 0)] + total_flux);

        //minimum of total, horizontal, and vertical flux
        min_cell_volume = 
            MIN(volume[THARR2D(0, 0, 0)] + total_flux,
            MIN(volume[THARR2D(0, 0, 0)] + right_flux - left_flux,
                volume[THARR2D(0, 0, 0)] + top_flux - bottom_flux));

        if (volume_change <= 0.0)
        {
            err_condition_shared[threadIdx.x] = 1;
        }
        if (min_cell_volume <= 0.0)
        {
            err_condition_shared[threadIdx.x] = 2;
        }

        recip_volume = 1.0/volume[THARR2D(0, 0, 0)];

        energy_change = ((pressure[THARR2D(0, 0, 0)] / density0[THARR2D(0, 0, 0)])
            + (viscosity[THARR2D(0, 0, 0)] / density0[THARR2D(0, 0, 0)]))
            * total_flux * recip_volume;

        energy1[THARR2D(0, 0, 0)] = energy0[THARR2D(0, 0, 0)] - energy_change;
        density1[THARR2D(0, 0, 0)] = density0[THARR2D(0, 0, 0)] * volume_change;
    }

    Reduce< BLOCK_SZ/2 >::run(err_condition_shared, error_condition, max_func);
}

__global__ void device_PdV_cuda_kernel_not_predict
(int x_min, int x_max, int y_min, int y_max, double dt,
         int * __restrict const error_condition,
const double * __restrict const xarea,
const double * __restrict const yarea,
const double * __restrict const volume,
const double * __restrict const density0,
      double * __restrict const density1,
const double * __restrict const energy0,
      double * __restrict const energy1,
const double * __restrict const pressure,
const double * __restrict const viscosity,
const double * __restrict const xvel0,
const double * __restrict const yvel0,
const double * __restrict const xvel1,
const double * __restrict const yvel1)
{
    __kernel_indexes;

    __shared__ int err_condition_shared[BLOCK_SZ];
    err_condition_shared[threadIdx.x] = 0;

    double volume_change;
    double recip_volume, energy_change, min_cell_volume,
        right_flux, left_flux, top_flux, bottom_flux, total_flux;
    
    if (row >= (y_min + 1) && row <= (y_max + 1)
    && column >= (x_min + 1) && column <= (x_max + 1))
    {
        left_flux   = (xarea[THARR2D(0, 0, 1)]
            * (xvel0[THARR2D(0, 0, 1)] + xvel0[THARR2D(0, 1, 1)] 
            + xvel1[THARR2D(0, 0, 1)] + xvel1[THARR2D(0, 1, 1)]))
            * 0.25 * dt;
        right_flux  = (xarea[THARR2D(1, 0, 1)]
            * (xvel0[THARR2D(1, 0, 1)] + xvel0[THARR2D(1, 1, 1)] 
            + xvel1[THARR2D(1, 0, 1)] + xvel1[THARR2D(1, 1, 1)]))
            * 0.25 * dt;

        bottom_flux = (yarea[THARR2D(0, 0, 0)]
            * (yvel0[THARR2D(0, 0, 1)] + yvel0[THARR2D(1, 0, 1)] 
            + yvel1[THARR2D(0, 0, 1)] + yvel1[THARR2D(1, 0, 1)]))
            * 0.25 * dt;
        top_flux    = (yarea[THARR2D(0, 1, 0)]
            * (yvel0[THARR2D(0, 1, 1)] + yvel0[THARR2D(1, 1, 1)] 
            + yvel1[THARR2D(0, 1, 1)] + yvel1[THARR2D(1, 1, 1)]))
            * 0.25 * dt;

        total_flux = right_flux - left_flux + top_flux - bottom_flux;

        volume_change = volume[THARR2D(0, 0, 0)]
            / (volume[THARR2D(0, 0, 0)] + total_flux);

        min_cell_volume =
            MIN(volume[THARR2D(0, 0, 0)] + total_flux,
            MIN(volume[THARR2D(0, 0, 0)] + right_flux - left_flux,
                volume[THARR2D(0, 0, 0)] + top_flux - bottom_flux));

        if (volume_change <= 0.0)
        {
            err_condition_shared[threadIdx.x] = 1;
        }
        if (min_cell_volume <= 0.0)
        {
            err_condition_shared[threadIdx.x] = 2;
        }

        recip_volume = 1.0/volume[THARR2D(0, 0, 0)];

        energy_change = ((pressure[THARR2D(0, 0, 0)] / density0[THARR2D(0, 0, 0)])
            + (viscosity[THARR2D(0, 0, 0)] / density0[THARR2D(0, 0, 0)]))
            * total_flux * recip_volume;

        energy1[THARR2D(0, 0, 0)] = energy0[THARR2D(0, 0, 0)] - energy_change;
        density1[THARR2D(0, 0, 0)] = density0[THARR2D(0, 0, 0)] * volume_change;
    }

    Reduce< BLOCK_SZ/2 >::run(err_condition_shared, error_condition, max_func);
}

extern "C" void pdv_kernel_cuda_
(int *error_condition, int *prdct, double *dtbyt)
{
    chunk.PdV_kernel(error_condition, *prdct, *dtbyt);
}

void CloverleafCudaChunk::PdV_kernel
(int* error_condition, int predict, double dt)
{
    CUDA_BEGIN_PROFILE;

    if (predict)
    {
        device_PdV_cuda_kernel_predict<<< num_blocks, BLOCK_SZ >>>
        (x_min, x_max, y_min, y_max, dt, pdv_reduce_array,
            xarea, yarea, volume, density0, density1,
            energy0, energy1, pressure, viscosity,
            xvel0, yvel0, xvel1, yvel1);
        CUDA_ERR_CHECK;
    }
    else
    {
        device_PdV_cuda_kernel_not_predict<<< num_blocks, BLOCK_SZ >>>
        (x_min, x_max, y_min, y_max, dt, pdv_reduce_array,
            xarea, yarea, volume, density0, density1,
            energy0, energy1, pressure, viscosity,
            xvel0, yvel0, xvel1, yvel1);
        CUDA_ERR_CHECK;
    }

    *error_condition = *thrust::max_element(reduce_pdv,
                                            reduce_pdv + num_blocks);

    CUDA_END_PROFILE;

    if (1 == *error_condition)
    {
        std::cerr << "Negative volume in PdV kernel" << std::endl;
    }
    else if (2 == *error_condition)
    {
        std::cerr << "Negative cell volume in PdV kernel" << std::endl;
    }
}

