
#include "chunk_cuda.cu"

#include "cuda_common.cu"

CloverleafCudaChunk chunk;

extern "C" void initialise_cuda_
(INITIALISE_ARGS)
{
    chunk = CloverleafCudaChunk(
        in_x_min,
        in_x_max,
        in_y_min,
        in_y_max,

        in_left,
        in_right,
        in_top,
        in_bottom,
        in_left_boundary,
        in_right_boundary,
        in_top_boundary,
        in_bottom_boundary,
        in_task);
}

CloverleafCudaChunk::CloverleafCudaChunk
(INITIALISE_ARGS)
:x_min(*in_x_min),
x_max(*in_x_max),
y_min(*in_y_min),
y_max(*in_y_max),
left(*in_left),
right(*in_right),
top(*in_top),
bottom(*in_bottom),
left_boundary(*in_left_boundary),
right_boundary(*in_right_boundary),
top_boundary(*in_top_boundary),
bottom_boundary(*in_bottom_boundary),
task(*in_task),
num_blocks((((*in_x_max)+4)*((*in_y_max)+4))/BLOCK_SZ)
{
    #define CUDA_ARRAY_ALLOC(arr, size)                              \
        hipMalloc((void**) &arr, size);                            \
        hipDeviceSynchronize();   \
        errChk(__LINE__, __FILE__);

    CUDA_ARRAY_ALLOC(volume, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(soundspeed, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(pressure, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(viscosity, BUFSZ2D(0, 0));

    CUDA_ARRAY_ALLOC(density0, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(density1, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(energy0, BUFSZ2D(0, 0));
    CUDA_ARRAY_ALLOC(energy1, BUFSZ2D(0, 0));

    CUDA_ARRAY_ALLOC(xvel0, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(xvel1, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(yvel0, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(yvel1, BUFSZ2D(1, 1));

    CUDA_ARRAY_ALLOC(xarea, BUFSZ2D(1, 0));
    CUDA_ARRAY_ALLOC(vol_flux_x, BUFSZ2D(1, 0));
    CUDA_ARRAY_ALLOC(mass_flux_x, BUFSZ2D(1, 0));

    CUDA_ARRAY_ALLOC(yarea, BUFSZ2D(0, 1));
    CUDA_ARRAY_ALLOC(vol_flux_y, BUFSZ2D(0, 1));
    CUDA_ARRAY_ALLOC(mass_flux_y, BUFSZ2D(0, 1));

    CUDA_ARRAY_ALLOC(cellx, BUFSZX(0));
    CUDA_ARRAY_ALLOC(celldx, BUFSZX(0));
    CUDA_ARRAY_ALLOC(vertexx, BUFSZX(1));
    CUDA_ARRAY_ALLOC(vertexdx, BUFSZX(1));

    CUDA_ARRAY_ALLOC(celly, BUFSZY(0));
    CUDA_ARRAY_ALLOC(celldy, BUFSZY(0));
    CUDA_ARRAY_ALLOC(vertexy, BUFSZY(1));
    CUDA_ARRAY_ALLOC(vertexdy, BUFSZY(1));

    CUDA_ARRAY_ALLOC(work_array_1, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(work_array_2, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(work_array_3, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(work_array_4, BUFSZ2D(1, 1));
    CUDA_ARRAY_ALLOC(work_array_5, BUFSZ2D(1, 1));

    reduce_ptr_1 = thrust::device_ptr< double >(work_array_1);
    reduce_ptr_2 = thrust::device_ptr< double >(work_array_2);
    reduce_ptr_3 = thrust::device_ptr< double >(work_array_3);
    reduce_ptr_4 = thrust::device_ptr< double >(work_array_4);
    reduce_ptr_5 = thrust::device_ptr< double >(work_array_5);

    CUDA_ARRAY_ALLOC(pdv_reduce_array, num_blocks*sizeof(int));
    reduce_pdv = thrust::device_ptr< int >(pdv_reduce_array);

    thr_cellx = thrust::device_ptr< double >(cellx);
    thr_celly = thrust::device_ptr< double >(celly);
    thr_xvel0 = thrust::device_ptr< double >(xvel0);
    thr_yvel0 = thrust::device_ptr< double >(yvel0);
    thr_xvel1 = thrust::device_ptr< double >(xvel1);
    thr_yvel1 = thrust::device_ptr< double >(yvel1);
    thr_density0 = thrust::device_ptr< double >(density0);
    thr_energy0 = thrust::device_ptr< double >(energy0);
    thr_pressure = thrust::device_ptr< double >(pressure);
    thr_soundspeed = thrust::device_ptr< double >(soundspeed);

    // TODO initialise to depth instead of 3
    CUDA_ARRAY_ALLOC(dev_left_buffer, sizeof(double)*y_max*4);
    CUDA_ARRAY_ALLOC(dev_right_buffer, sizeof(double)*y_max*4);
    CUDA_ARRAY_ALLOC(dev_top_buffer, sizeof(double)*x_max*4);
    CUDA_ARRAY_ALLOC(dev_bottom_buffer, sizeof(double)*x_max*4);
}

