#include "hip/hip_runtime.h"

#include "ftocmacros.h"
#include "cuda_common.cu"

#include "chunk_cuda.cu"
extern CloverleafCudaChunk chunk;

__global__ void device_reset_field_kernel_cuda
(int x_min,int x_max,int y_min,int y_max,
      double* __restrict const density0,
const double* __restrict const density1,
      double* __restrict const energy0,
const double* __restrict const energy1,
      double* __restrict const xvel0,
const double* __restrict const xvel1,
      double* __restrict const yvel0,
const double* __restrict const yvel1)
{
    __kernel_indexes;

    if(row >= (y_min + 1) && row <= (y_max + 1) + 1
    && column >= (x_min + 1) && column <= (x_max + 1) + 1)
    {
        xvel0[THARR2D(0, 0, 1)] = xvel1[THARR2D(0, 0, 1)];
        yvel0[THARR2D(0, 0, 1)] = yvel1[THARR2D(0, 0, 1)];

        if(row <= (y_max + 1)
        && column <= (x_max + 1))
        {
            density0[THARR2D(0, 0, 0)] = density1[THARR2D(0, 0, 0)];
            energy0[THARR2D(0, 0, 0)]  = energy1[THARR2D(0, 0, 0)];
        }
    }
}

extern "C" void reset_field_kernel_cuda_
(int *x_min,int *x_max,int *y_min,int *y_max,
      double* density0,
const double* density1,
      double* energy0,
const double* energy1,
      double* xvel0,
const double* xvel1,
      double* yvel0,
const double* yvel1)
{
    chunk.reset_field_kernel();
}

void CloverleafCudaChunk::reset_field_kernel
(void)
{
_CUDA_BEGIN_PROFILE_name(device);
    device_reset_field_kernel_cuda<<< num_blocks, BLOCK_SZ >>>
    (x_min,x_max,y_min,y_max, density0, density1,
        energy0, energy1, xvel0, xvel1, yvel0, yvel1);
    errChk(__LINE__, __FILE__);
_CUDA_END_PROFILE_name(device);
}
